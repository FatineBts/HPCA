#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hiprand/hiprand.h> //bibliothèque pour la generation de nombres aléatoires  
#include <hiprand/hiprand_kernel.h>

#define OUTPUT_FILE "/tmp/3302011/absorbed.dat"
#define NbpaquetN 512

char info[] = "\
Usage:\n\
    neutron-gpu H Nb C_c C_s\n\
\n\
    H  : épaisseur de la plaque\n\
    Nb : nombre d'échantillons\n\
    C_c: composante absorbante\n\
    C_s: componente diffusante\n\
\n\
Exemple d'execution : \n\
    neutron-gpu 1.0 500000000 0.5 0.5\n\
";

/*
 * générateur uniforme de nombres aléatoires dans l'intervalle [0,1)
 */

static int iDivUp(int a, int b){ //donne la division de a par b
  return ((a % b != 0) ? (a / b + 1) : (a / b));
}

/*
 * notre gettimeofday()
 */
double my_gettimeofday(){
  struct timeval tmp_time;
  gettimeofday(&tmp_time, NULL);
  return tmp_time.tv_sec + (tmp_time.tv_usec* 1.0e-6L);
}

__global__ void setup_kernel(hiprandState* state, unsigned long seed)
{
    int id_thread = threadIdx.x + blockIdx.x*blockDim.x;
    //threadIx.x = numéro du thread qui va de 0 à la dimension du bloc -1
    //blockId = va correspondre au numéro du bloc qu'on considère 
    //blockDim = dimension du bloc (nombre de cases dans une direction donc si on considère qu'une case = 1 thread, donne le nbr de threads par bloc)
    //gridDim = donne le nombre de blocs dans une grille (n/NbTheadsParBloc)
    //gridDim*blockDim = nombre de threads dans une grille = n

    //va donner par exemple : 0+0*5 (0),..,4+0*5 (4, dernier element du premier bloc),0+1*5 (5),...,4+1*5 (9, dernier element du second bloc)
    hiprand_init(seed, id_thread, 0, &state[id_thread]);
}

/*
  Fonction qui va servir à générer des nombres différents pour chaque thread. 
*/

__device__ float generate(hiprandState* globalState, int ind) 
{
    hiprandState localState = globalState[ind];
    float random = hiprand_uniform(&localState);
    globalState[ind] = localState;
    return random;
}

__global__ void kernel(hiprandState* globalState, float* absorbed, float h, int n, float c, float c_c, float c_s, int paquetN, int* result) //uniquement les elements qui sont intialisés dans le main + r b et t
{
  float d; 
  float x; 
  float L;
  float u;
  int i = blockDim.x*blockIdx.x + threadIdx.x; //sert de compteur
  int gi = i;
  int prev;
  __shared__ int r_local[NbpaquetN]; //taille du nombre de threads qu'il va nous falloir pour traiter paquetN neutrons par thread, comme on a imposé 512 threads par bloc dans le main, NbpaquetN vaudra 512, on aura donc 512 threads qui vont traiter chacun 1 paquetN donc 512 paquetN. Il s'agit du nombre de paquetN.  
  __shared__ int t_local[NbpaquetN]; 
  __shared__ int b_local[NbpaquetN];
  r_local[threadIdx.x] = 0; //on initialise à zéro le tableau
  t_local[threadIdx.x] = 0; 
  b_local[threadIdx.x] = 0; 
  //int r_updated = 0, t_updated = 0, b_updated = 0;  
  int r, t, b; 

  while(i<n){ //i doit s'incrémenter mais pas gi
  d = 0.0; 
  x = 0.0;
  while (1) { 
    u = generate(globalState,gi); 
    L = -(1 / c) * log(u);
    x = x + L * cos(d);
    if (x < 0) { //reflechi  
    r_local[threadIdx.x] = r_local[threadIdx.x] + 1;
     break;
    } 
    else if (x >= h) { //transmis 
    t_local[threadIdx.x] = t_local[threadIdx.x] + 1;
    break;
    } 
    else if ((u = generate(globalState,gi)) < c_c / c) { //absorbé
     b_local[threadIdx.x] = b_local[threadIdx.x] + 1; 
     prev = atomicAdd(result+3,1); //communication interphread pas possible donc on veut l'atomicAdd pour pas écrire de manière concurente (on donne la main à 1 thread) 
     absorbed[prev] = x;
      break;
    } 
    else {
      u = generate(globalState,gi);
      d = u * M_PI; //direction
    } 
  } //boucle while(1)
  i += (gridDim.x*blockDim.x); //nombre de threads dans une grille qui correspond ici à un bloc, on fait des sauts correspondants aux nombres de threads dans un bloc ce qui donne 512 
}//while(i<n) //tant qu'on a pas traité tous les neutrons 

 r_local[threadIdx.x] = r; //on initialise à zéro le tableau
 t_local[threadIdx.x] = t; 
 b_local[threadIdx.x] = b; 
  __syncthreads(); //synchronize the local threads writing to the local memory cache 

  int j = blockDim.x / 2; 

  while(j>0)
  {
    if(threadIdx.x < j)
    {
      r_local[threadIdx.x]+=r_local[threadIdx.x+j];
      t_local[threadIdx.x]+=t_local[threadIdx.x+j];
      b_local[threadIdx.x]+=b_local[threadIdx.x+j];
    }
    j/=2; 
    __syncthreads();
  }

if(threadIdx.x == 0){//le premier thread va faire les calculs 
  atomicAdd(result,r_local[0]); 
  atomicAdd(result+1,b_local[0]);
  atomicAdd(result+2,t_local[0]);
}//fin if 

}

int main(int argc, char *argv[]) {

  // chronometrage
  double start, finish;

  if( argc == 1)
    fprintf( stderr, "%s\n", info);

  float c, c_c, c_s;
  float h;
  int r, b, t;
  int n,j; 
  int paquetN;
  
    // valeurs par defaut
  h = 1.0;
  n = 500000000;
  c_c = 0.5;
  c_s = 0.5;

  // recuperation des parametres
  if (argc > 1)
    h = atof(argv[1]);
  if (argc > 2)
    n = atoi(argv[2]);
  if (argc > 3)
    c_c = atof(argv[3]);
  if (argc > 4)
    c_s = atof(argv[4]);
  if (argc > 5)
     paquetN = atof(argv[5]);
  c = c_s + c_c; 
  r = b = t = j = 0;

  // affichage des parametres pour verificatrion
  printf("Épaisseur de la plaque : %4.g\n", h);
  printf("Nombre d'échantillons  : %d\n", n);
  printf("C_c : %g\n", c_c);
  printf("C_s : %g\n", c_s);
  float* absorbed_CPU;
  float* absorbed_GPU;
  int* result_CPU; 
  int* result_GPU;
  //il s'agit du nombre de neutrons traités par 1 thread = nombres de neutrons dans un paquet. 
  hiprandState* devStates;
  printf("paquetN : %d\n", paquetN);

  /* Définition du nombre de threads et de la taille de la grille */
  dim3 NbThreadsParBloc(NbpaquetN,1,1); dim3 NbBlocks;  

  NbBlocks.x = NbpaquetN;//iDivUp(iDivUp(n,paquetN),NbThreadsParBloc.x); //on fait en sorte qu'au lieu qu'un thread traite 1 neutron, 1 thread va en traiter paquetN. On impose le nombre de threads par blocs à 512 et on cherche le nombre de blocs qu'il faudrait si on a n neutrons avec un paquet de neutrons traité par 1 thread égal à paquetN. Plus on augmente paquetN et plus n est petit et à priori plus la vitesse d'execution devrait être élevée.   
  

  //printf("n/paquetN %d\n",iDivUp(n,paquetN));
  printf("nombre de blocs %4.2d\n",NbBlocks.x);
  NbBlocks.y = 1;
  NbBlocks.z = 1;  

  /* Allocation de la mémoire */
  absorbed_CPU = (float *) calloc(n,sizeof(float)); //sur CPU 
  result_CPU = (int *) calloc(4,sizeof(int)); //sur CPU
  hipMalloc((void**) &absorbed_GPU, n*sizeof(float)); //sur GPU
  hipMalloc (&devStates, NbThreadsParBloc.x*NbBlocks.x*sizeof(hiprandState));
  hipMalloc((void**) &result_GPU, 4*sizeof(int));

  hipMemcpy(absorbed_CPU, absorbed_GPU, n*sizeof(float), hipMemcpyHostToDevice); //copie du absorbed CPU dans GPU 
  hipMemcpy(result_CPU, result_GPU, 4*sizeof(int), hipMemcpyHostToDevice);

  // debut du chronometrage
  start = my_gettimeofday();


  #pragma omp parallel 
  {
  #pragma omp master
  {
  setup_kernel <<<NbBlocks,NbThreadsParBloc>>> (devStates,unsigned(time(NULL)));  //initialisation de l'état curandState pour chaque thread
  kernel<<<NbBlocks, NbThreadsParBloc>>>(devStates, absorbed_GPU, h, n, c, c_c, c_s, paquetN, result_GPU); //génération des positions absorbed pour GPU
   //on renvoie aussi r, t et b pour l'affichage plus loin dans le code 
  
  hipMemcpy(absorbed_CPU, absorbed_GPU, n*sizeof(float), hipMemcpyDeviceToHost); //copie du absorbed GPU dans CPU 
  hipMemcpy(result_CPU, result_GPU, 4*sizeof(int), hipMemcpyDeviceToHost);


 }//fin omp master 

  }//fin du pragma omp parallel


  // fin du chronometrage
  finish = my_gettimeofday();

  r = result_CPU[0]; 
  b = result_CPU[1]; 
  t = result_CPU[2]; 
  j = result_CPU[3];

  printf("\nPourcentage des neutrons refléchis : %4.2g\n", (float) r / (float) n);
  printf("Pourcentage des neutrons absorbés : %4.2g\n", (float) b / (float) n);
  printf("Pourcentage des neutrons transmis : %4.2g\n", (float) t / (float) n);

  printf("\nTemps total de calcul: %.8g sec\n", finish - start);
  printf("Millions de neutrons /s: %.2g\n", (double) n / ((finish - start)*1e6));

  printf("réfléchis = %d, absorbés = %d, transmis = %d\n", r, b,t);
  printf("Total traité: %d\n", r + b +t);

/*
  // ouverture du fichier pour ecrire les positions des neutrons absorbés
  FILE *f_handle = fopen(OUTPUT_FILE, "w");
  if (!f_handle) {
     fprintf(stderr, "Cannot open " OUTPUT_FILE "\n");
     exit(EXIT_FAILURE);
 }
  
  for (j = 0; j < b; j++)
     fprintf(f_handle, "%f\n", absorbed_CPU[j]);
  
  // fermeture du fichier
  fclose(f_handle);
  printf("Result written in " OUTPUT_FILE "\n"); 
*/

  hipFree(absorbed_GPU); 
  hipFree(devStates);
  hipFree(result_GPU);
  free(result_CPU); 
  free(absorbed_CPU);


  return EXIT_SUCCESS;
}
